#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <queue>
#include <cmath>
#include <hip/hip_runtime.h>
#include <pcl/point_cloud.h>
#include <pcl/point_types.h>
#include <opencv2/opencv.hpp>

#define eChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int MAX_DIM = 3;   // Maximum dimensions of points (can be changed)
const int N_POINTS = 1e4, N_QUERIES = 1e6, K_NEIGHBORS = 5, INF = 1e9, RANGE_MAX = 100, N_PRINT = 10;

struct Point {
    float coords[MAX_DIM];  // Coordinates in MAX_DIM-dimensional space
};

struct KDNode {
    Point point;
    int left;
    int right;
    int axis;  // Splitting axis (0 to MAX_DIM-1)
};

// Function Prototypes
__host__ void printPoints(Point *points, int n);
__host__ void generatePoints(Point *points, int n);
__host__ void buildKDTree(Point *points, KDNode *tree, int n, int m);
__global__ void kNearestNeighborsGPU(KDNode *tree, int treeSize, Point *queries, Point *results, int nQueries, int k);
__host__ void printResults(Point *queries, Point *results, int start, int end);


// Function to convert depth map to PCL point cloud
pcl::PointCloud<pcl::PointXYZ>::Ptr depthMapToPointCloud(const cv::Mat& depthMap, const cv::Mat& cameraMatrix) {
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud(new pcl::PointCloud<pcl::PointXYZ>());

    float fx = cameraMatrix.at<float>(0, 0); // focal length x
    float fy = cameraMatrix.at<float>(1, 1); // focal length y
    float cx = cameraMatrix.at<float>(0, 2); // principal point x
    float cy = cameraMatrix.at<float>(1, 2); // principal point y

    for (int y = 0; y < depthMap.rows; y++) {
        for (int x = 0; x < depthMap.cols; x++) {
            float depth = depthMap.at<float>(y, x);

            if (depth > 0) {
                pcl::PointXYZ point;
                point.x = (x - cx) * depth / fx;
                point.y = (y - cy) * depth / fy;
                point.z = depth;
                cloud->points.push_back(point);
            }
        }
    }
    cloud->width = (int)cloud->points.size();
    cloud->height = 1;
    cloud->is_dense = false;
    return cloud;
}


// CUDA kernel to convert pcl::PointXYZ to Point
__global__ void convertPointCloudKernel(const pcl::PointXYZ* inputPoints, Point* outputPoints, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        outputPoints[idx].coords[0] = inputPoints[idx].x;  // x coordinate
        outputPoints[idx].coords[1] = inputPoints[idx].y;  // y coordinate
        outputPoints[idx].coords[2] = inputPoints[idx].z;  // z coordinate
    }
}

void savePointCloudToStructCUDA(pcl::PointCloud<pcl::PointXYZ>::Ptr cloud, Point* points) {
    int numPoints = cloud->size();

    // Allocate host memory for the output points

    // Allocate device memory
    pcl::PointXYZ* deviceInputPoints;
    Point* deviceOutputPoints;

    hipMalloc((void**)&deviceInputPoints, numPoints * sizeof(pcl::PointXYZ));
    hipMalloc((void**)&deviceOutputPoints, numPoints * sizeof(Point));

    // Copy input point cloud data to the device
    hipMemcpy(deviceInputPoints, cloud->points.data(), numPoints * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = 256;
    int gridSize = (numPoints + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    convertPointCloudKernel<<<gridSize, blockSize>>>(deviceInputPoints, deviceOutputPoints, numPoints);

    // Copy the results back to the host
    hipMemcpy(points, deviceOutputPoints, numPoints * sizeof(Point), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceInputPoints);
    hipFree(deviceOutputPoints);
}


int main() {
    srand(16);

    int TREE_SIZE = 1;
    while (TREE_SIZE < N_POINTS) TREE_SIZE <<= 1;

    Point *points;
    KDNode *tree;
    Point *queries;

    eChk(hipMallocManaged(&points, N_POINTS * sizeof(Point)));
    eChk(hipMallocManaged(&tree, TREE_SIZE * sizeof(KDNode)));
    eChk(hipMallocManaged(&queries, N_QUERIES * sizeof(Point)));

    generatePoints(points, N_POINTS);
    buildKDTree(points, tree, N_POINTS, TREE_SIZE);
    generatePoints(queries, N_QUERIES);

    auto start = std::chrono::system_clock::now();

    Point *results;
    eChk(hipMallocManaged(&results, N_QUERIES * K_NEIGHBORS * sizeof(Point)));

    kNearestNeighborsGPU<<<32768, 32>>>(tree, TREE_SIZE, queries, results, N_QUERIES, K_NEIGHBORS);
    eChk(hipDeviceSynchronize());
    
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0 * std::chrono::duration<float>(end - start).count();

    printResults(queries, results, N_QUERIES - N_PRINT - 1, N_QUERIES);

    std::cout << "Elapsed time in milliseconds : " << duration << "ms\n\n";

    eChk(hipFree(results));
    eChk(hipFree(points));
    eChk(hipFree(tree));
    eChk(hipFree(queries));
}

// Helper function to generate random points in MAX_DIM dimensions
__host__ void generatePoints(Point *points, int n) {
    for (int i = 0; i < n; i++) {
        for (int d = 0; d < MAX_DIM; d++) {
            points[i].coords[d] = static_cast<float>(rand() % RANGE_MAX + 1);
        }
    }
}

// Comparator for sorting points based on the current axis
struct PointComparator {
    int axis;
    PointComparator(int ax) : axis(ax) {}

    bool operator()(const Point &p1, const Point &p2) {
        return p1.coords[axis] < p2.coords[axis];
    }
};

// Recursive function to build KDTree
__host__ void buildSubTree(Point *points, KDNode *tree, int start, int end, int depth, int node) {
    if (start >= end) return;

    int axis = depth % MAX_DIM;
    std::sort(points + start, points + end, PointComparator(axis));

    int split = (start + end - 1) / 2;
    tree[node].point = points[split];
    tree[node].axis = axis;

    buildSubTree(points, tree, start, split, depth + 1, node * 2);
    buildSubTree(points, tree, split + 1, end, depth + 1, node * 2 + 1);
}

// Function to initialize the KDTree
__host__ void buildKDTree(Point *points, KDNode *tree, int n, int treeSize) {
    for (int i = 0; i < treeSize; i++) {
        tree[i].left = tree[i].right = -1;  // Default values
    }
    buildSubTree(points, tree, 0, n, 0, 1);
}

// Device function to compute Euclidean distance between two points
__device__ float distance(const Point &p1, const Point &p2) {
    float dist = 0.0f;
    for (int i = 0; i < MAX_DIM; i++) {
        dist += powf(p1.coords[i] - p2.coords[i], 2);
    }
    return sqrtf(dist);
}

// Device function to compare two points for K-nearest neighbor search
struct KNNComparator {
    __device__ bool operator()(const Point &p1, const Point &p2, const Point &query) {
        return distance(p1, query) < distance(p2, query);
    }
};

// Recursive device function for finding K nearest neighbors
__device__ void findKNearestNeighbors(KDNode *tree, int treeSize, int treeNode, int depth, Point query, Point *neighbors, int k) {
    // Base case
    if (treeNode >= treeSize) return;

    KDNode node = tree[treeNode];
    if (node.axis == -1) return;

    // Push the current node point into neighbors array
    for (int i = 0; i < k; i++) {
        if (i == k - 1 || distance(node.point, query) < distance(neighbors[i], query)) {
            neighbors[i] = node.point;
            break;
        }
    }

    // Find the next subtree to search
    int nextAxis = (depth + 1) % MAX_DIM;
    if (query.coords[node.axis] < node.point.coords[node.axis]) {
        findKNearestNeighbors(tree, treeSize, treeNode * 2, depth + 1, query, neighbors, k);
    } else {
        findKNearestNeighbors(tree, treeSize, treeNode * 2 + 1, depth + 1, query, neighbors, k);
    }
}

// Kernel to perform K nearest neighbor search for all queries
__global__ void kNearestNeighborsGPU(KDNode *tree, int treeSize, Point *queries, Point *results, int nQueries, int k) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nQueries) {
        Point neighbors[K_NEIGHBORS];  // Local array to store the K nearest neighbors
        findKNearestNeighbors(tree, treeSize, 1, 0, queries[index], neighbors, k);

        // Copy the neighbors to the results array
        for (int i = 0; i < k; i++) {
            results[index * k + i] = neighbors[i];
        }
    }
}

// Print a list of points
void printPoints(Point *points, int n) {
    for (int i = 0; i < n; i++) {
        std::cout << "[";
        for (int d = 0; d < MAX_DIM; d++) {
            std::cout << points[i].coords[d];
            if (d < MAX_DIM - 1) std::cout << ", ";
        }
        std::cout << "] ";
    }
    std::cout << std::endl;
}

// Print query results
__host__ void printResults(Point *queries, Point *results, int start, int end) {
    for (int i = start; i < end; i++) {
        std::cout << "Query: [";
        for (int d = 0; d < MAX_DIM; d++) {
            std::cout << queries[i].coords[d];
            if (d < MAX_DIM - 1) std::cout << ", ";
        }
        std::cout << "]\n";

        for (int j = 0; j < K_NEIGHBORS; j++) {
            std::cout << "\tNeighbor " << j+1 << ": [";
            for (int d = 0; d < MAX_DIM; d++) {
                std::cout << results[i * K_NEIGHBORS + j].coords[d];
                if (d < MAX_DIM - 1) std::cout << ", ";
            }
            std::cout << "]\n";
        }
        std::cout << std::endl;
    }
}
